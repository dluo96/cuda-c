// Matrix multiplication kernel where each thread computes one element of the output matrix.
// Assumptions: inputs are square matrices. 



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <cassert>

__global__ void matMul(int *a, int *b, int *c, int N) {
    // Every thread computes one element in the output matrix.
    // Calculate global row and column for each thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Boundary check for our matrix
    if (row < N && col < N) {

        // As every thread computes one element in the output matrix `c`,
        // every thread traverses one row in `a` and one column in `b`. 
        int tmp = 0;
        for (int k = 0; k < N; k++) {
            tmp += a[row * N + k] * b[k * N + col];
        }

        // Write element to output matrix
        c[row * N + col] = tmp;
    }
}

// Initialise a square matrix with random numbers between 0-100
void init_matrix(int *m, int N)
{
    for (int i = 0; i < N * N; i++) {
        m[i] = rand() % 100;
    }
}

// Verify the result of the GPU kernel with a CPU calculation
void verify_result(int *a, int *b, int *c, int N)
{
    int tmp;
    // For every row
    for(int i = 0; i < N; i++) {
        // For every column
        for(int j = 0; j < N; j++) {
            // For every element in the row-col pair ij
            tmp = 0;
            for(int k = 0; k < N; k++) {
                tmp += a[i * N + k] * b[k * N + j];
            }

            // Check each element
            assert(tmp == c[i * N + j]);
        }
    }
}

int main() {
    // Set square matrix dimensions (2^10 x 2^10 here)
    int N = 1 << 10;
    size_t bytes = N * N * sizeof(int);

    // Allocate host memory
    int *a, *b, *c;

    // Allocate unified memory, which is accessible from the CPU or the GPU - with `cudaMallocManaged` 
    // (rather than `cudaMalloc`), the CUDA runtime manages the transfer of memory back and forth for 
    // you, so memcpy isn't needed
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    // Initialise our matrices
    init_matrix(a, N);
    init_matrix(b, N);

    // Make a 2D grid (16x16) of blocks, where each block is itself a 2D grid of threads (16x16).
    // This makes the indexing of the output matrix more intuitive. We use the `dim3` CUDA type to do this.
    // Note that we need to calculate the number of blocks needed in each dimension to cover the entire 
    // output matrix of size N x N.
    int threadsPerBlockDim = 16;
    int blocksPerGridDim = (N + threadsPerBlockDim - 1) / threadsPerBlockDim;
    dim3 dimBlock(threadsPerBlockDim, threadsPerBlockDim);
    dim3 dimGrid(blocksPerGridDim, blocksPerGridDim);

    // Invoke matrix multiplication kernel: every launched thread
    // will calculate one element of the resulting matrix
    matMul<<<dimGrid, dimBlock>>>(a, b, c, N);

    // As we are not doing a Memcpy (a synchronizing operation) due to using `cudaMallocManaged`,
    // we need synchronize explicitly:
    hipDeviceSynchronize();

    // Verify the result
    verify_result(a, b, c, N);

    printf("Success! Computed matrix multiplication where each thread computes one element of the output matrix.\n");
    return 0;
}


