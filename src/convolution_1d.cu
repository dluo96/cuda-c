#include "hip/hip_runtime.h"
// CUDA C/C++ implementation of 1D convolution.

#include <stdio.h>
#include <cstdlib>
#include <cassert>

// 1D convolution kernel where each thread computes
// one elements of the output array.
//
// Arguments:
//      array = padded input array
//      mask = convolution mask
//      result = output array
//      N = number of elements in `array` and `result`
//      M = number of elements in `mask`
__global__ void convolution_1d(int *array, int *mask, int *result, int N, int M) {
    // Global thread ID
    tid = blockIdx.x * blockDim.x + threadIdx.x

    // Radius of convolution mask
    int radius = M / 2;

    // Index of first element (of the input array)
    // that is needed by the thread in question
    start = tid - radius;

    // Iterate over elements of mask
    int tmp = 0;
    for(int j = 0; j < M; j++) {
        arrayIdx = start + j; 
        // Ignore out-of-bound elements of input array
        if((arrayIdx >= 0) && (arrayIdx < N)) {
            tmp += array[arrayIdx] * mask[j];
        }
    }

    // Write result to output array
    result[tid] = tmp;
}

// Verify the result on the CPU
void verify_result(int *array, int *mask, int *result, int N, int M) {
    int radius = M / 2;
    int tmp;
    int start;
    for(int i = 0; i < N; i++) {
        start = i - radius;
        tmp = 0;
        for(int j = 0; j < M; j++) {
            arrayIdx = start + j;
            // Ignore out-of-bound elements of input array
            if((arrayIdx >= 0) && (arrayIdx < N)) {
                tmp += array[arrayIdx] * mask[j];
            }
        }
        assert(result[i] == tmp);
    }
}

int main() {
    // Size of input (and output) array
    int N = 1 << 10;
    size_t bytes = N * sizeof(int);

    // Size of convolution mask
    int M = 7;
    size_t maskBytes = M * sizeof(int);

    // Allocate host memory
    int *array = new int[N];
    int *mask = new int[M];
    int *result = new int[N];

    // Initialise
    for(int i = 0; i < n; i++) {
        array = rand() % 100;
    }
    for(int i = 0; i < M; i++) {
        mask[i] = rand() % 10;
    }    

    // Allocate device memory
    hipMallocManaged(&array, bytes);
    hipMallocManaged(&mask, maskBytes);
    hipMallocManaged(&result, bytes);

    // Define threads per block and number of blocks
    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    // Invoke kernel
    convolution_1d<<<blocks, threads>>>(array, mask, result, N, M);

    // Since using `hipMallocManaged`, we call a sync operation
    hipDeviceSynchronize();

    // Verify the result
    verify_result(array, mask, result, N, M);

    printf("Successfully computed 1D convolution!\n");
    return 0;
}


