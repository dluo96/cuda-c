#include <iostream>
#include <cassert>
#include "sum_reduction.h"

void init_vector(int *a, int N) {
    for (int i = 0; i < N; i++) {
        a[i] = 1;  // Initializing with 1 for simplicity
    }
}

int main() {
        int N = 1 << 24;
        size_t bytes = N * sizeof(int);

        int *input, *result;
        int *d_input, *d_result;

        // Allocate CPU and GPU memory, populate input, and copy to device
        input = (int*)malloc(bytes);
        result = (int*)malloc(bytes);
        hipMalloc(&d_input, bytes);
        hipMalloc(&d_result, bytes);
        init_vector(input, N);
        hipMemcpy(d_input, input, bytes, hipMemcpyHostToDevice);

        // Block size (#threads) and grid size (#blocks)
        int blockSize = SHMEM_LEN;
        int gridSize = (N/2 + blockSize - 1) / blockSize; // Division by 2 is for v4-v5

        // CUDA events for timing kernels
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float milliseconds = 0;

        // Perform first kernel call
        hipEventRecord(start);
        sum_reduction_v4<<<gridSize, blockSize>>>(d_input, d_result, N);

        // Track how many partial results are left to be added and perform kernel 
        // decomposition with recursion.
        // Note: although CUDA kernel launches are asynchronous, all GPU-related tasks
        // placed in one stream (the default behavior) are executed sequentially. 
        // Hence there is no need for `hipDeviceSynchronize` between kernel calls here.
        unsigned int numRemain = gridSize;
        while(numRemain > 1) {
                gridSize = (numRemain/2 + blockSize - 1) / blockSize; // Division 2 is for v4-v5
                sum_reduction_v4<<<gridSize, blockSize>>>(d_result, d_result, numRemain);
                numRemain = gridSize;
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);

        // Copy to host
        hipMemcpy(result, d_result, bytes, hipMemcpyDeviceToHost);

        // Check result
        printf("Result: %d \n", result[0]);
        assert(result[0] == N);
        printf("Success! Computed sum reduction.\n");
        printf("Time elapsed: %f milliseconds\n", milliseconds);

        // Cleanup
        hipFree(d_input);
        hipFree(d_result);
        free(input);
        free(result);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        return 0;
}
